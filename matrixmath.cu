/*************************************************************************************************
 * File: matrixmath.cu
 * Date: 11/06/2018
 * 
 * Compiling: Requires a Nvidia CUDA capable graphics card and the Nvidia GPU Computing Toolkit.
 *            Linux & Windows: nvcc -Wno-deprecated-gpu-targets -O3 -o prog2 matrixmath.cu
 *          
 * Usage:   Linux: >> prog2
 *          Windows: PS > ./prog2.exe
 * 
 * Description: This file runs a parallel program using CUDA to find the sum of squares. The first 
 *      part of the program asks whether you would like to run the optimized completely parallel 
 *      solution or an equivalent sequential solution. Both solutions use CUDA, but 1 is optimized 
 *      to be ran on many cores using atomic addition while the other runs the entire calculation 
 *      on a single pass-through, similar to how a sequential CPU program would run. Once the type 
 *      of kernel to run has been decided the user is asked how large they would like the sum of 
 *      squares to calculate. This calculation is done by creating an NxN matrix and a N sized vector.
 *      The matrix (A) and the vector (B) create a new vector C that satisfies the following formula:
 *      C[i] += A[i][j] * B[j]
 *
 *************************************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>

#define GRIDVAL 20.0

/**
 * __global__ void matrixSum(int, int, int, int, int)
 * - Function is a __global__ function meaning it is accessible for GPGPU processing.
 *   The function takes in a NxN matrix as *a, and a N length vector *b and an empty 
 *   N length vector *c along with the N value (both l and w are N in this case). The
 *   function calculates c[x] += a[x][y] * b[y] and performs an atomicAdd function when
 *   adding into c[x]. This function is meant to be highly parallelized.
 **/
__global__ void matrixSum(int *a, int *b, int *c, int l, int w) {
    // grab x position on grid
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    // grab y position on grid
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // safety check + math run
    if( x >= 0 && y >= 0 && x < w && y < l) {
        // perform c[x] += a[y][x] * b[y] using an atomic add
        atomicAdd(&c[x], a[(x*w)+y] * b[y]);
    }
}

/**
 * __global__ void singleSum(int, int, int ,int ,int)
 * - Function is a __global__ function meaning it is accessible for GPGPU processing.
 *   The function takes in a NxN matrix as *a, and a N length vector *b and an empty
 *   N length vector *c along with the N value (both l and w are N in this case). The
 *   function loops through each y value and each x value calculating 
 *   c[x] += a[x][y] * b[y]. The function is meant to run on a single CUDA core and is
 *   meant to represent a sequential run of the matrixSum function
 **/
__global__ void singleSum(int *a, int *b, int *c, int l, int w) {
    // loop through all y values
    for(int i = 0; i < w; i++) {
        // loop through all x values
        for(int j = 0; j < l; j++) {
            // perform c[i] += a[y][x] * b[x]
            c[i] += a[(i*w)+j]*b[j];
        }
    }
}

/**
 * int main(int, char*[])
 * - Function is the entry point for the program. Welcomes the user, then asks the user 
 *   whether they want to run a sequential or parallel calculation for the sum of squares.
 *   Once a selection is made the program asks the user for the max square to use (also 
 *   known as the size N for the NxN matrix and N length vectors). When both these values
 *   have been entered then the NxN matrix and N length vectors are allocated and initialized
 *   with their starting values, the function then calls the external __global__ function 
 *   with the appropriate grid/block set-up and returns the result out.
 */
int main(int argc, char* argv[]) {

    // declare a size variable and a sequential flag
    int size, sequential;

    // give a hello prompt and prompt for either sequential or parallel
    std::cout << "Sum of Squares using CUDA." << std::endl;
    std::cout << "Enter 1 for Sequential calculation or enter 0 for Parallel calculation: ";
    std::cin >> sequential;
    
    // let the user know the selection they just made
    if (sequential == 1) 
        std::cout << "SEQUENTIAL calculation is ON." << std::endl << std::endl;
    else 
        std::cout << "PARALLEL calculation is ON." << std::endl << std::endl;
    
    // prompt user for N value of the matrix and vector
    std::cout << "Enter in the maximum square to calculate: ";
    std::cin >> size;

    // prepare a NxN matrix, and two N length vectors and populate them with valid data
    int *a = new int[size*size];
    int *b = new int[size];
    int *c = new int[size];
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            a[(i*size)+j] = j+1;
        }
        b[i] = i+1;
        c[i] = 0;
    }

    // declare 3 variables that will be used on the GPU
    int *gpu_a, *gpu_b, *gpu_c;

    // allocate space on the GPU for the incoming matrix and vectors
    hipMalloc( (void**)&gpu_a, (size * size)*sizeof(int));
    hipMalloc( (void**)&gpu_b, (size)*sizeof(int));
    hipMalloc( (void**)&gpu_c, (size)*sizeof(int));

    // copy all the matrix and vector data to the GPU, set gpu_c to be all 0s
    hipMemcpy(gpu_a, a, size*size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(gpu_c, 0, size*sizeof(int));

    // create a dim3 go find the number of blocks and number of threads per block given the user's input size
    // and the staticly defined GRIDVAL variable
    dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
    dim3 numBlocks(ceil(size/GRIDVAL), ceil(size/GRIDVAL), 1);

    // if we are running the sequential program, run the singleSum function with 1 block and 1 thread
    if (sequential == 1) 
        singleSum<<<1, 1>>>(gpu_a, gpu_b, gpu_c, size, size);
    // if w are running the parallel program, run the matrixSum function with the previously calculated num of blocks & threads
    else
        matrixSum<<<numBlocks, threadsPerBlock>>>(gpu_a, gpu_b, gpu_c, size, size);
    
    // copy the results from the GPGPU computation back to the CPU
    hipMemcpy(c, gpu_c, size*sizeof(int), hipMemcpyDeviceToHost);

    // output the result of C
    printf("Resulting values of the vector C:\n");
    for(int i = 0; i < size; i++) {
        printf("%d | ", c[i]);
    }
    printf("\n");

    //return a 0 for successful program run.
    return 0;
}
